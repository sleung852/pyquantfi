#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdexcept>
#include <algorithm>
#include <math.h>

// declare function to be ran in the GPU
__global__ void monte_carlo_sim(double* d_St, const double* d_Zs, const double* d_drift, const double* d_sigma,
	const double* d_deltaT, const int* d_n, const int* d_m) {

	// get thread, block ids and block dim
    const unsigned tid = threadIdx.x;
    const unsigned bid = blockIdx.x;
    const unsigned bsz = blockDim.x;

    // create stock_index
    int idx = tid + bid * bsz;

    // for each simluated stock
    if (idx < *d_m) {
		int t = 0;
		while (t < *d_n) {
    		// compute stock price from previous stock price 
    		d_St[idx] = d_St[idx] * (*d_drift) * exp((*d_sigma) * sqrt(*d_deltaT) * d_Zs[idx*(*d_n) + t]);
			t++;
    	}
    }
}

// function to setup running Monte Carlo Simulation in the GPU
void run_mc_cuda(double* ST, double S, double sigma, double rate, double T, int n,
	int m, double* Zs) {

	// assign values in the CPU
	double deltaT = T/n;
	double drift = exp((rate - 0.5*(pow(sigma, 2.0)))*T/n);
	for (size_t i=0; i<m; i++) ST[i] = S; 

	// declarations for variables in the GPU
	//array
	double* d_St = nullptr;
	double* d_Zs = nullptr;
	// double and int
	double* d_sigma = nullptr;
	double* d_deltaT = nullptr;
	double* d_drift = nullptr;
	int* d_n = nullptr;
	int* d_m = nullptr;

	// memory allocations in GPU
	// array
	hipMalloc((void **) &d_Zs, n*m*sizeof(double));
	hipMalloc((void **) &d_St, m*sizeof(double));
	// double & int
	hipMalloc(&d_sigma, sizeof(double));
	hipMalloc(&d_deltaT, sizeof(double));
	hipMalloc(&d_drift, sizeof(double));
	hipMalloc(&d_n, sizeof(int));
	hipMalloc(&d_m, sizeof(int));

	// copy values from host (CPU) to device (GPU)
	// array
	hipMemcpy(d_Zs, Zs, n*m*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_St, ST, m*sizeof(double), hipMemcpyHostToDevice);
	// double & int
	hipMemcpy(d_sigma, &sigma, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_deltaT, &deltaT, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_drift, &drift, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_m, &m, sizeof(int), hipMemcpyHostToDevice);

	// Use either 128 or 256 for THREADS_PER_BLOCK
	int TPB = 1024;
	
	// invoke the Monte Carlo kernel
	monte_carlo_sim<<<(m+TPB-1)/TPB,TPB>>>(d_St, d_Zs, d_drift, d_sigma, d_deltaT, d_n, d_m);

	// copy the results from the device (GPU) back to the host (CPU)
	hipMemcpy(ST, d_St, m*sizeof(double), hipMemcpyDeviceToHost);

	// free device memory
	hipFree(d_Zs);
	hipFree(d_St);
	hipFree(d_sigma);
	hipFree(d_deltaT);
	hipFree(d_drift);
	hipFree(d_n);
	hipFree(d_m);
}
