#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdexcept>
#include <algorithm>
#include <math.h>

// declare function to be ran in the GPU
__global__ void monte_carlo_sim(double* d_ST, const double* d_Zs,
	const double* d_S, const double* d_drift, const double* d_sigma,
	const double* d_deltaT, const int* d_n, const int* d_m);

// function to setup running Monte Carlo Simulation in the GPU
void run_mc(double* ST, double S, double sigma, double rate, double T, int n,
	int m, double* Zs) {

	// declare ptr values in the CPU
	double* h_S;
	double* h_sigma;
	double* h_deltaT;
	double* h_drift;
	int* h_n;
	int* h_m;

	// create ptr values in the hash
	h_S = new double;
	h_sigma = new double;
	h_deltaT = new double;
	h_drift = new double;
	h_n = new int;
	h_m = new int;

	// assign values in the CPU
	*h_S = S;
	*h_sigma = sigma;
	*h_deltaT = T/n;
	*h_drift = exp(rate - 0.5*(pow(sigma, 2.0))*T/n);
	*h_n = n;
	*h_m = m;

	// declarations for variables in the GPU
	//array
	double* d_ST = nullptr;
	double* d_Zs = nullptr;
	// double and int
	double* d_S;
	double* d_sigma;
	double* d_deltaT;
	double* d_drift;
	int* d_n;
	int* d_m;

	// memory allocations in GPU
	// array
	hipMalloc((void**)&d_Zs, n*m*sizeof(double));
	hipMalloc((void**)&d_ST, m * sizeof(double));
	// double & int
	hipMalloc(&d_S, sizeof(double));
	hipMalloc(&d_sigma, sizeof(double));
	hipMalloc(&d_deltaT, sizeof(double));
	hipMalloc(&d_drift, sizeof(double));
	hipMalloc(&d_n, sizeof(int));
	hipMalloc(&d_m, sizeof(int));

	// copy values from host (CPU) to device (GPU)
	// array
	hipMemcpy(d_Zs, Zs, n*m*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_ST, ST, m*sizeof(double), hipMemcpyHostToDevice);
	// double & int
	hipMemcpy(d_S, h_S, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_sigma, h_sigma, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_deltaT, h_deltaT, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_drift, h_drift, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_n, h_n, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_m, h_m, sizeof(int), hipMemcpyHostToDevice);

	// calculate the threads and blocks for the GPU
	int threadsPerBlock,blocksPerGrid;
	if (n*m<1024){
		threadsPerBlock = n*m;
		blocksPerGrid   = 1;
	} else {
		threadsPerBlock = 1024;
		blocksPerGrid   = ceil(double(n*m)/double(threadsPerBlock));
	}

	// invoke the Monte Carlo kernel
	monte_carlo_sim<<<blocksPerGrid,threadsPerBlock>>>(d_ST, d_Zs,
		d_S, d_drift, d_sigma, d_deltaT, d_n, d_m);

	// copy the results from the device (GPU) back to the host (CPU)
	hipMemcpy(ST, d_ST, m*sizeof(double), hipMemcpyDeviceToHost);

	// free device memory
	hipFree(d_Zs);
	hipFree(d_ST);
	hipFree(d_S);
	hipFree(d_sigma);
	hipFree(d_deltaT);
	hipFree(d_drift);
	hipFree(d_n);
	hipFree(d_m);

	// free host memory
	delete[] Zs;
	delete h_S;
	delete h_sigma;
	delete h_deltaT;
	delete h_drift;
	delete h_n;
	delete h_m;
}


__global__ void monte_carlo_sim(double* d_ST, const double* d_Zs,
	const double* d_S, const double* d_drift, const double* d_sigma,
	const double* d_deltaT, const int* d_n, const int* d_m) {

	// get thread, block ids and block dim
    const unsigned tid = threadIdx.x;
    const unsigned bid = blockIdx.x;
    const unsigned bsz = blockDim.x;

    // create stock_index and time_index
    int s_idx = tid + bid * bsz;
    int n_idx = tid + bid * bsz;

    double S = *d_S;

    // for each simluated stock
    if (s_idx < *d_m) {
    	int ni = 0;
    	do {
    		// compute stock price from previous stock price 
    		S = S * *d_drift * exp(*d_sigma * sqrt(*d_deltaT) * d_Zs[n_idx]);
    	}
    	// while time has not reached T
    	while (ni < *d_n);
    	d_ST[s_idx] = S;
    }
}